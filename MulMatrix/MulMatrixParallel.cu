#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <locale>

#include <stdio.h>

#define BLOCK_SIZE 16

hipError_t MulMatrixCuda(float* mul_matrix, float* mul_matrix2, float* matrix1, float * matrix2, int n);
void print_matrix(float* mtx, int n);

__global__ void mtxMult(float *C, float *A, float *B, int n)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	float sum = 0.0;

	int ia = n * BLOCK_SIZE * by + n * ty; // A[i,0] - ������ ������ (������� �������� � ������)
	int ib = BLOCK_SIZE * bx + tx;         // B[0,j] - ������ ������� (������� �������� �������)

	for (int k = 0; k < n; k++) // ���������� ��������
	{
		sum += A[ia + k] * B[ib + k * n];
	}

	int ic = n * BLOCK_SIZE*by + BLOCK_SIZE * bx; // ����� ������ ������� � ����� ����������
	C[ic + n * ty + tx] = sum; // ���������� ���������
}

__global__ void mtxMult2(float *C, float *A, float *B, int n)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int aBegin = n * BLOCK_SIZE * by;
	int aEnd = aBegin + n - 1;
	int bBegin = BLOCK_SIZE * bx;
	int aStep = BLOCK_SIZE;
	int bStep = BLOCK_SIZE * n;

	float sum = 0.0f;

	for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep)
	{
		__shared__ float as[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float bs[BLOCK_SIZE][BLOCK_SIZE];

		as[ty][tx] = A[ia + n * ty + tx];
		bs[ty][tx] = B[ib + n * ty + tx];
		__syncthreads();    // ������ ���������������� (���������� ��������� ���������)

		for (int k = 0; k < BLOCK_SIZE; k++)
			sum += as[ty][k] * bs[k][tx];

		__syncthreads(); // ���������� ������ �� �����
	}

	C[n * BLOCK_SIZE * by + BLOCK_SIZE * bx + n * ty + tx] = sum;
}

int main()
{
	setlocale(LC_ALL, "Russian");

	const int k = 400;
	const int n = k * BLOCK_SIZE; // ����������� �������, ������� BLOCK_SIZE

	/*float matrix1[n*n] = { 0 };
	float matrix2[n*n] = { 0 };
	float mul_matrix[n*n] = { 0 };
	float mul_matrix2[n*n] = { 0 };*/

	float* matrix1;
	matrix1 = new float[n*n];
	float * matrix2;
	matrix2 = new float[n*n];
	float * mul_matrix;
	mul_matrix = new float[n*n];
	float * mul_matrix2;
	mul_matrix2 = new float[n*n];


	// ������������� ������
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			matrix1[n * i + j] = i * 10 + j;
		}
	}

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			matrix2[n *i + j] = i + 10 * j; // (i == j) ? 1 : 0; ��� ��������� �������
		}
	}


	// ����� ������� �� �������
	print_matrix(matrix1, n);
	print_matrix(matrix2, n);
	printf("\n\n");

	hipError_t cudaStatus = MulMatrixCuda(mul_matrix, mul_matrix2, matrix1, matrix2, n);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// ����� ������� �� �������
	print_matrix(mul_matrix, n);
	print_matrix(mul_matrix2, n);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t MulMatrixCuda(float* mul_matrix, float* mul_matrix2, float* matrix1, float * matrix2, int n)
{
	int numBytes = n * n * sizeof(float);
	float *dev_matrix1 = 0;
	float *dev_matrix2 = 0;
	float *dev_mul_matrix = 0;
	float *dev_mul_matrix2 = 0;
	hipError_t cudaStatus;


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

#pragma region ��������� ������ � DRAM
	cudaStatus = hipMalloc((void**)&dev_mul_matrix, numBytes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_mul_matrix2, numBytes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_matrix1, numBytes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_matrix2, numBytes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
#pragma endregion // ��� ��� ������: 1-��, 2-�� � ��������������

#pragma region ����������� ������ �� CPU � DRAM
	cudaStatus = hipMemcpy(dev_matrix1, matrix1, numBytes, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_matrix2, matrix2, numBytes, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
#pragma endregion // ��� ���� ��������� ������

	dim3 blocks(n / BLOCK_SIZE, n / BLOCK_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	//������ GPU
	hipEvent_t start, stop; // ���������� ����������
	float elapsedTimeInMs = 0;
	hipEventCreate(&start); // �������������
	hipEventCreate(&stop);  // �������������
	hipEventRecord(start, 0); // ������ �������

// ������ ����
	mtxMult << <blocks, threads >> > (dev_mul_matrix, dev_matrix1, dev_matrix2, n);

	hipEventRecord(stop, 0); // ��������� �������
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTimeInMs, start, stop);
	printf("����������� ����� 1-�� ������ GPU: %.8f ��\n\n", elapsedTimeInMs);


	hipEventCreate(&start); // �������������
	hipEventCreate(&stop);  // �������������
	hipEventRecord(start, 0); // ������ �������

	mtxMult2 << <blocks, threads >> > (dev_mul_matrix2, dev_matrix1, dev_matrix2, n);

	hipEventRecord(stop, 0); // ��������� �������
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTimeInMs, start, stop);
	printf("����������� ����� 2-�� ������ GPU: %.8f ��\n\n", elapsedTimeInMs);



	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(mul_matrix, dev_mul_matrix, numBytes, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(mul_matrix2, dev_mul_matrix2, numBytes, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_mul_matrix);
	hipFree(dev_mul_matrix2);
	hipFree(dev_matrix1);
	hipFree(dev_matrix2);

	return cudaStatus;
}

void print_matrix(float* mtx, int n)
{
	return;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			float m = mtx[n * i + j];
			printf("%6g|", m);
		}
		printf("\n");
	}
	printf("\n");
}