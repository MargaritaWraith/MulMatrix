#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <locale>

#include <stdio.h>

hipError_t MulMatrixCuda(double **mul_matrix, double **matrix1, double **matrix2, unsigned int m1, unsigned int n1, unsigned int m2, unsigned int n2);

__global__ void mulKernel(double **mul_matrix, double **matrix1, double **matrix2)
{
	int i = threadIdx.x;



	
}

int main()
{
	setlocale(LC_ALL, "Russian");

	const int m1 = 5;
	const int n1 = 9;
	const int m2 = 9;
	const int n2 = 5;

	double matrix1[m1][n1] = { 0 };
	double matrix2[m2][n2] = { 0 };
	double mul_matrix[m1][n2] = { 0 };
	//double **test = &mul_matrix;

	if (n1 != m2)
	{
		printf("����������� ������ �� ���������");
		return 0;
	}

	for (int i = 0; i < m1; i++)
	{
		for (int j = 0; j < n1; j++)
		{
			matrix1[i][j] = 10 * i + j;
		}
	}

	for (int i = 0; i < m1; i++)
	{
		for (int j = 0; j < n1; j++)
		{
			matrix2[i][j] = i + 10 * j;
		}
	}


	hipError_t cudaStatus = MulMatrixCuda((double**)mul_matrix, (double**)matrix1, (double**)matrix2, m1, n1, m2, n2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	/*printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);*/

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t MulMatrixCuda(double **mul_matrix, double **matrix1, double **matrix2,
	unsigned int m1, unsigned int n1, unsigned int m2, unsigned int n2)
{
	double *dev_matrix1 = 0;
	double *dev_matrix2 = 0;
	double *dev_mul_matrix = 0;
	hipError_t cudaStatus;


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	cudaStatus = hipMalloc((void**)&dev_mul_matrix, m1 * n2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_matrix1, m1 * n1 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_matrix2, m2 * n2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input matrix from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_matrix1, matrix1, m1*n1 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_matrix2, matrix2, m2*n2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	mulKernel << <1, size >> > (dev_mul_matrix, dev_matrix1, dev_matrix2);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(mul_matrix, dev_mul_matrix, m1*n2 * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_mul_matrix);
	hipFree(dev_matrix1);
	hipFree(dev_matrix2);

	return cudaStatus;
}
